#include "hip/hip_runtime.h"
// Copyright 2019 ETH Zürich, Thomas Schöps
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its contributors
//    may be used to endorse or promote products derived from this software
//    without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include <hipcub/hipcub.hpp>
#include <libvis/cuda/cuda_auto_tuner.h>

#include "badslam/cost_function.cuh"
#include "badslam/cuda_util.cuh"
#include "badslam/cuda_matrix.cuh"
#include "badslam/surfel_projection_nvcc_only.cuh"
#include "badslam/util.cuh"
#include "badslam/util_nvcc_only.cuh"

namespace vis {

__global__ void ResetSurfelAccumCUDAKernel(
    u32 surfels_size,
    CUDABuffer_<float> surfels,
    CUDABuffer_<u8> active_surfels) {
  const unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfels_size) {
    if (!(active_surfels(0, surfel_index) & kSurfelActiveFlag)) {
      return;
    }
    // 10.30
    /*surfels(kSurfelAccum0, surfel_index) = 0;
    surfels(kSurfelAccum1, surfel_index) = 0;
    surfels(kSurfelAccum2, surfel_index) = 0;
    surfels(kSurfelAccum3, surfel_index) = 0;
    surfels(kSurfelAccum4, surfel_index) = 0;
    surfels(kSurfelAccum5, surfel_index) = 0;
    surfels(kSurfelAccum6, surfel_index) = 0;
    surfels(kSurfelAccum7, surfel_index) = 0;
    surfels(kSurfelAccum8, surfel_index) = 0;*/
    constexpr int kSurfelAccumuArr[35] = {12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33,34,35,36,37,38,39,40,41,42,43,44,45,46};
    for (int i = 0; i < 35; ++i){
      surfels(kSurfelAccumuArr[i], surfel_index) = 0;
    }
  }
}

void CallResetSurfelAccumCUDAKernel(
    hipStream_t stream,
    u32 surfels_size,
    const CUDABuffer_<float>& surfels,
    const CUDABuffer_<u8>& active_surfels) {
  CUDA_AUTO_TUNE_1D(
      ResetSurfelAccumCUDAKernel,
      512,
      surfels_size,
      0, stream,
      /* kernel parameters */
      surfels_size,
      surfels,
      active_surfels);
  CUDA_CHECK();
}


__global__ void ResetSurfelAccum0to3CUDAKernel(
    u32 surfels_size,
    CUDABuffer_<float> surfels,
    CUDABuffer_<u8> active_surfels) {
  const unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfels_size) {
    if (!(active_surfels(0, surfel_index) & kSurfelActiveFlag)) {
      return;
    }
    
    surfels(kSurfelAccum0, surfel_index) = 0;
    surfels(kSurfelAccum1, surfel_index) = 0;
    surfels(kSurfelAccum2, surfel_index) = 0;
    surfels(kSurfelAccum3, surfel_index) = 0;
  }
}

void CallResetSurfelAccum0to3CUDAKernel(
    hipStream_t stream,
    u32 surfels_size,
    const CUDABuffer_<float>& surfels,
    const CUDABuffer_<u8>& active_surfels) {
  CUDA_AUTO_TUNE_1D(
      ResetSurfelAccum0to3CUDAKernel,
      512,
      surfels_size,
      0, stream,
      /* kernel parameters */
      surfels_size,
      surfels,
      active_surfels);
  CUDA_CHECK();
}


template<bool use_depth_residuals>
__global__ void AccumulateSurfelPositionAndDescriptorOptimizationCoeffsCUDAKernel(
    SurfelProjectionParameters s,
    PixelCenterUnprojector depth_unprojector,
    DepthToColorPixelCorner depth_to_color,
    PixelCornerProjector color_corner_projector,
    hipTextureObject_t color_texture,
    CUDABuffer_<u8> active_surfels) {
  const unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (surfel_index < s.surfels_size) {
    if (!(active_surfels(0, surfel_index) & kSurfelActiveFlag)) {
      return;
    }
    
    SurfelProjectionResult6 r;
    if (SurfelProjectsToAssociatedPixel(surfel_index, s, &r)) {
      float3 rn = s.frame_T_global.Rotate(r.surfel_normal);
      
      // --- Depth residual change wrt. position change ---
      if (use_depth_residuals) {
        float depth_residual_inv_stddev =
            ComputeDepthResidualInvStddevEstimate(depth_unprojector.nx(r.px), depth_unprojector.ny(r.py), r.pixel_calibrated_depth, rn, s.depth_params.baseline_fx);
        
        const float depth_jacobian = -depth_residual_inv_stddev;
        
        float3 local_unproj;
        float raw_depth_residual;
        ComputeRawDepthResidual(
            depth_unprojector, r.px, r.py, r.pixel_calibrated_depth,
            depth_residual_inv_stddev,
            r.surfel_local_position, rn, &local_unproj, &raw_depth_residual);
        
        const float depth_weight = ComputeDepthResidualWeight(raw_depth_residual);
        
        // Accumulate:
        // 10.29 shouldn't it be depth_weight^2? 
        // jzmTODO: 
        // s.surfels(kSurfelAccum0, surfel_index) += depth_weight * depth_weight * depth_jacobian * depth_jacobian;
        // s.surfels(kSurfelAccum0, surfel_index) += depth_weight * depth_jacobian * depth_jacobian;
        // s.surfels(kSurfelAccum6, surfel_index) += depth_weight * raw_depth_residual * depth_jacobian;
        s.surfels(kSurfelAccum0, surfel_index) += depth_weight * depth_jacobian * depth_jacobian;
        s.surfels(kSurfelAccum0 + 28, surfel_index) += depth_weight * raw_depth_residual * depth_jacobian;// 28 is the number of kAccums used for H, 28 = 6 * (6 + 1) / 2
        /*if (surfel_index == 0){
          printf("use depth residual \n");
        }*/
      }
      // --------------------------------------------------
      
      
      float2 color_pxy;
      if (TransformDepthToColorPixelCorner(r.pxy, depth_to_color, &color_pxy)) {
        // --- Descriptor residual ---
          /*if (surfel_index == 0){
            printf("surfel_index: %d \n", surfel_index);
            float x = tex2D<float4>(color_texture, color_pxy.x, color_pxy.y).x;
            float y = tex2D<float4>(color_texture, color_pxy.x, color_pxy.y).y;
            float z = tex2D<float4>(color_texture, color_pxy.x, color_pxy.y).z;
            printf("(%f, %f, %f) \n",x,y,z);
          }*/
        float2 t1_pxy, t2_pxy;
        ComputeTangentProjections(
            r.surfel_global_position,
            r.surfel_normal,
            SurfelGetRadiusSquared(s.surfels, surfel_index),
            s.frame_T_global,
            color_corner_projector,
            &t1_pxy,
            &t2_pxy);
        /*
        const float surfel_descriptor_1 = s.surfels(kSurfelDescriptor1, surfel_index);
        const float surfel_descriptor_2 = s.surfels(kSurfelDescriptor2, surfel_index);
        float raw_descriptor_residual_1;
        float raw_descriptor_residual_2;
        ComputeRawDescriptorResidual(
            color_texture, color_pxy, t1_pxy, t2_pxy, surfel_descriptor_1, surfel_descriptor_2, &raw_descriptor_residual_1, &raw_descriptor_residual_2);
        */
        constexpr int kSurfelDescriptorArr[6] = {6,7,8,9,10,11};
        float surfel_descriptor[6]; // problematic with const float array and use for loop to initialize
        for (int i = 0; i< 6; ++i){
          surfel_descriptor[i] = s.surfels(kSurfelDescriptorArr[i], surfel_index);
        }
        float raw_descriptor_residual[6];
        ComputeRawFeatureDescriptorResidual(
          color_texture, // TODO: use feature_texture
          color_pxy,
          t1_pxy,
          t2_pxy,
          surfel_descriptor,
          raw_descriptor_residual);
        // 10.30 these are consts for a given surfel
        const float term1 = -color_corner_projector.fx * (rn.x*r.surfel_local_position.z - rn.z*r.surfel_local_position.x);
        const float term2 = -color_corner_projector.fy * (rn.y*r.surfel_local_position.z - rn.z*r.surfel_local_position.y);
        const float term3 = 1.f / (r.surfel_local_position.z * r.surfel_local_position.z);
        // ---------------------------
        // Accumulate H and b
        // constexpr int kSurfelAccumuHArr[35] = {12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33,34,35,36,37,38,39}; // I think this is too large, so for now I just hard code by idx+12
        // constexpr int kSurfelAccumubArr[7] = {40,41,42,43,44,45,46};
        // 10.30 iterate over feature channels to accumulate H and b
          // loop over channel n in N:
          //    accumulate H(0,0)
          //    accumulate H(0,1+n)   for residual_1
          //    accumulate H(0,1+n+N) for residual_2
          //    accumulate H(n+1,n+1) for jacobian w.r.t. descirptor
          //    accumulate b(n)
        for (int channel_i = 1; channel_i < 4; ++channel_i){
          
          // --- Descriptor residual change wrt. position change ---
          // 10.30 gradients varies form channel to channel
          float grad_x_1;
          float grad_y_1;
          float grad_x_2;
          float grad_y_2;
          DescriptorJacobianWrtProjectedPositionOnChannels(color_texture, color_pxy, t1_pxy, t2_pxy, &grad_x_1, &grad_y_1, &grad_x_2, &grad_y_2, channel_i);
          
          float jacobian_wrt_position_1 = -(grad_x_1 * term1 + grad_y_1 * term2) * term3;
          float jacobian_wrt_position_2 = -(grad_x_2 * term1 + grad_y_2 * term2) * term3;
          // -------------------------------------------------------
        
          // --- Descriptor residual change wrt. descriptor change ---
          constexpr float jacobian_wrt_descriptor = -1.f;
          // ---------------------------------------------------------
          
          // --- Compute weights for each residual term ---
          const float weight_1 = ComputeDescriptorResidualWeight(raw_descriptor_residual[channel_i]); // 10.30 jzmTODO: maybe the weight needs adjusted when applying feature maps
          const float weighted_raw_residual_1 = weight_1 * raw_descriptor_residual[channel_i];
          
          const float weight_2 = ComputeDescriptorResidualWeight(raw_descriptor_residual[channel_i+3]);// N = 3
          const float weighted_raw_residual_2 = weight_2 * raw_descriptor_residual[channel_i+3];

          // Accumulate:
          // kSurfelAccum0: H(0, 0)
          // kSurfelAccum1: H(0, 1) = H(1, 0)
          // kSurfelAccum2: H(0, 2) = H(2, 0)
          // kSurfelAccum3: H(1, 1)
          // kSurfelAccum4: H(1, 2) = H(2, 1)
          // kSurfelAccum5: H(2, 2)
          // kSurfelAccum6: b(0)
          // kSurfelAccum7: b(1)
          // kSurfelAccum8: b(2)
          // Accumulate formula for H: The index of H_{ij} in kSurfelAccumuHArr satisfy idx = i*2N+j-i*(i-1)/2, since kSurfelAccumuHArr is too large, for now I hard code the indices for acessing surfel buffer: idx = i*2N+j-i*(i-1)/2 + 12
          // Since we only need to compute the diagonal of H, namely H(k,k), idx = k*2N+k-k*(k-1)/2, k = {0,1,2...,2N}
          // Accumulate formula for b: index of bi where i = 0,...,2N, b_0 is for depth residual and descriptor residual, b_1 - b_2N if for descriptor residual. 
          // The depth residual impacts only b_0.
          // ---------------------------------------------------------
          // We fill H rowwise, mainly deal with the H(0,:) and diag(H)
          // H(0,0)
          s.surfels(kSurfelAccum0, surfel_index) += weight_1 * jacobian_wrt_position_1 * jacobian_wrt_position_1 +
                                                  weight_2 * jacobian_wrt_position_2 * jacobian_wrt_position_2;  // from residual 2
          // H(0,channel_i), from residual_1 H(0,0) reserves a spot and channel_i starts from 1 to 3
          s.surfels(kSurfelAccum0 + channel_i, surfel_index) += weight_1 * jacobian_wrt_position_1 * jacobian_wrt_descriptor;
          // H(0,1+channel_i+N), from residual_2
          s.surfels(kSurfelAccum0 + channel_i + 3, surfel_index) += weight_2 * jacobian_wrt_position_2 * jacobian_wrt_descriptor;
          // H(k,k)
          // jzmTODO:check int 
          s.surfels(kSurfelAccum0 + int(channel_i*6+channel_i-channel_i*(channel_i-1)/2), surfel_index) += weight_1 * jacobian_wrt_descriptor * jacobian_wrt_descriptor; // from residual_1
          s.surfels(kSurfelAccum0 + int((channel_i+3)*6+(channel_i+3)-(channel_i+3)*((channel_i+3)-1)/2), surfel_index) += weight_2 * jacobian_wrt_descriptor * jacobian_wrt_descriptor; // from residual_2
          // In total, there are 28 unique non-zero entries in H, start of b: kSurfelAccum0 + 28
          // accumulate b_0, depth residual related part is already accumulated
          s.surfels(kSurfelAccum0  + 28, surfel_index) += weighted_raw_residual_1 * jacobian_wrt_position_1 + 
                                                    weighted_raw_residual_2 * jacobian_wrt_position_2;  // from residual 2
          s.surfels(kSurfelAccum0 + 28 + channel_i, surfel_index) += weighted_raw_residual_1 * jacobian_wrt_descriptor;
          s.surfels(kSurfelAccum0 + 28 + channel_i + 3, surfel_index) += weighted_raw_residual_2 * jacobian_wrt_descriptor;

        }
        
        
        
        
        
       
        /*const float weight_1 = ComputeDescriptorResidualWeight(raw_descriptor_residual_1);
        const float weighted_raw_residual_1 = weight_1 * raw_descriptor_residual_1;
        
        const float weight_2 = ComputeDescriptorResidualWeight(raw_descriptor_residual_2);
        const float weighted_raw_residual_2 = weight_2 * raw_descriptor_residual_2;*/
        /*
        if (surfel_index == 0){
          printf("weighted_raw_residual_1: %f\n",weighted_raw_residual_1);
        }
        */
        
        // Residual 1 (and some parts of 2, where accumulating onto the same variable)
        // jzmTODO: for diagnal entries of H, the weights should be squared.
        // s.surfels(kSurfelAccum0, surfel_index) += weight_1 * weight_1 * jacobian_wrt_position_1 * jacobian_wrt_position_1 +
        // weight_2 * jacobian_wrt_position_2 * jacobian_wrt_position_2; 
        // s.surfels(kSurfelAccum3, surfel_index) += weight_1 * weight_1 * jacobian_wrt_descriptor * jacobian_wrt_descriptor;
        // s.surfels(kSurfelAccum5, surfel_index) += weight_2 * weight_2 * jacobian_wrt_descriptor * jacobian_wrt_descriptor;
        /*s.surfels(kSurfelAccum0, surfel_index) += weight_1 * jacobian_wrt_position_1 * jacobian_wrt_position_1 +
                                                  weight_2 * jacobian_wrt_position_2 * jacobian_wrt_position_2;  // from residual 2
        s.surfels(kSurfelAccum1, surfel_index) += weight_1 * jacobian_wrt_position_1 * jacobian_wrt_descriptor;
        s.surfels(kSurfelAccum3, surfel_index) += weight_1 * jacobian_wrt_descriptor * jacobian_wrt_descriptor;
        s.surfels(kSurfelAccum6, surfel_index) += weighted_raw_residual_1 * jacobian_wrt_position_1 +
                                                  weighted_raw_residual_2 * jacobian_wrt_position_2;  // from residual 2
        s.surfels(kSurfelAccum7, surfel_index) += weighted_raw_residual_1 * jacobian_wrt_descriptor;
        
        // Residual 2
        s.surfels(kSurfelAccum2, surfel_index) += weight_2 * jacobian_wrt_position_2 * jacobian_wrt_descriptor;
        s.surfels(kSurfelAccum5, surfel_index) += weight_2 * jacobian_wrt_descriptor * jacobian_wrt_descriptor;
        s.surfels(kSurfelAccum8, surfel_index) += weighted_raw_residual_2 * jacobian_wrt_descriptor;*/
      }
    }
  }
}

void AccumulateSurfelPositionAndDescriptorOptimizationCoeffsCUDAKernel(
    hipStream_t stream,
    const SurfelProjectionParameters& s,
    const PixelCenterUnprojector& depth_unprojector,
    const DepthToColorPixelCorner& depth_to_color,
    const PixelCornerProjector& color_corner_projector,
    hipTextureObject_t color_texture,
    const CUDABuffer_<u8>& active_surfels,
    bool use_depth_residuals) {
  if (use_depth_residuals) {
    CUDA_AUTO_TUNE_1D(
        AccumulateSurfelPositionAndDescriptorOptimizationCoeffsCUDAKernel<true>,
        512,
        s.surfels_size,
        0, stream,
        /* kernel parameters */
        s,
        depth_unprojector,
        depth_to_color,
        color_corner_projector,
        color_texture,
        active_surfels);
  } else {
    CUDA_AUTO_TUNE_1D(
        AccumulateSurfelPositionAndDescriptorOptimizationCoeffsCUDAKernel<false>,
        512,
        s.surfels_size,
        0, stream,
        /* kernel parameters */
        s,
        depth_unprojector,
        depth_to_color,
        color_corner_projector,
        color_texture,
        active_surfels);
  }
  CUDA_CHECK();
}


__global__ void UpdateSurfelPositionAndDescriptorCUDAKernel(
    u32 surfels_size,
    CUDABuffer_<float> surfels,
    CUDABuffer_<u8> active_surfels) {
  const unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfels_size) {
    if (!(active_surfels(0, surfel_index) & kSurfelActiveFlag)) {
      return;
    }
    
    float H_0_0 = surfels(kSurfelAccum0, surfel_index);
    float H_0_1 = surfels(kSurfelAccum1, surfel_index);
    float H_0_2 = surfels(kSurfelAccum2, surfel_index);
    float H_1_1 = surfels(kSurfelAccum3, surfel_index);
    float H_1_2 = surfels(kSurfelAccum4, surfel_index);
    float H_2_2 = surfels(kSurfelAccum5, surfel_index);
    
    // Make sure that the matrix is positive definite
    // (instead of only semi-positive definite).
    constexpr float kEpsilon = 1e-6f;
    H_0_0 += kEpsilon;
    H_1_1 += kEpsilon;
    H_2_2 += kEpsilon;
    
    // Perform in-place Cholesky decomposition of H
    H_0_0 = sqrtf(H_0_0);
    H_0_1 = H_0_1 / H_0_0;
    H_1_1 = sqrtf(H_1_1 - H_0_1 * H_0_1);
    H_0_2 = H_0_2 / H_0_0;
    H_1_2 = (H_1_2 - H_0_2 * H_0_1) / H_1_1;
    H_2_2 = sqrtf(H_2_2 - H_0_2 * H_0_2 - H_1_2 * H_1_2);
    
    // Solve H * x = b for x.
    //
    // (H_0_0     0     0)   (H_0_0 H_0_1 H_0_2)   (x0)   (b0)
    // (H_0_1 H_1_1     0) * (    0 H_1_1 H_1_2) * (x1) = (b1)
    // (H_0_2 H_1_2 H_2_2)   (    0     0 H_2_2)   (x2)   (b2)
    //
    // Naming the result of the second multiplication y, we get:
    //
    // (H_0_0     0     0)   (y0)   (b0)
    // (H_0_1 H_1_1     0) * (y1) = (b1)
    // (H_0_2 H_1_2 H_2_2)   (y2)   (b2)
    // 
    // and:
    // 
    // (H_0_0 H_0_1 H_0_2)   (x0)   (y0)
    // (    0 H_1_1 H_1_2) * (x1) = (y1)
    // (    0     0 H_2_2)   (x2) = (y2)
    
    const float b0 = surfels(kSurfelAccum6, surfel_index);
    const float b1 = surfels(kSurfelAccum7, surfel_index);
    const float b2 = surfels(kSurfelAccum8, surfel_index);
    
    float y0 = b0 / H_0_0;
    float y1 = (b1 - H_0_1 * y0) / H_1_1;
    float y2 = (b2 - H_0_2 * y0 - H_1_2 * y1) / H_2_2;
    
    float x2 = y2 / H_2_2;
    float x1 = (y1 - H_1_2 * x2) / H_1_1;
    float x0 = (y0 - H_0_2 * x2 - H_0_1 * x1) / H_0_0;
    
    if (x0 != 0) {
      // Update surfel position
      float3 global_position = SurfelGetPosition(surfels, surfel_index);
      float3 surfel_normal = SurfelGetNormal(surfels, surfel_index);
      SurfelSetPosition(&surfels, surfel_index, global_position - x0 * surfel_normal);
    }
    
    if (x1 != 0) {
      float surfel_descriptor_1 = surfels(kSurfelDescriptor1, surfel_index);
      surfel_descriptor_1 -= x1;
      surfels(kSurfelDescriptor1, surfel_index) = ::max(-180.f, ::min(180.f, surfel_descriptor_1));
    }
    
    if (x2 != 0) {
      float surfel_descriptor_2 = surfels(kSurfelDescriptor2, surfel_index);
      surfel_descriptor_2 -= x2;
      surfels(kSurfelDescriptor2, surfel_index) = ::max(-180.f, ::min(180.f, surfel_descriptor_2));
    }
    
    // Reset accum fields for normal optimization.
    // surfels(kSurfelAccum0, surfel_index) = 0;
    // surfels(kSurfelAccum1, surfel_index) = 0;
    // surfels(kSurfelAccum2, surfel_index) = 0;
    // surfels(kSurfelAccum3, surfel_index) = 0;
  }
}

void CallUpdateSurfelPositionAndDescriptorCUDAKernel(
    hipStream_t stream,
    u32 surfels_size,
    CUDABuffer_<float> surfels,
    CUDABuffer_<u8> active_surfels) {
  CUDA_AUTO_TUNE_1D(
      UpdateSurfelPositionAndDescriptorCUDAKernel,
      512,
      surfels_size,
      0, stream,
      /* kernel parameters */  
      surfels_size,
      surfels,
      active_surfels);
  CUDA_CHECK();
}


__global__ void ResetSurfelAccum0to1CUDAKernel(
    u32 surfels_size,
    CUDABuffer_<float> surfels,
    CUDABuffer_<u8> active_surfels) {
  const unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfels_size) {
    if (!(active_surfels(0, surfel_index) & kSurfelActiveFlag)) {
      return;
    }
    
    surfels(kSurfelAccum0, surfel_index) = 0;
    surfels(kSurfelAccum1, surfel_index) = 0;
  }
}

void CallResetSurfelAccum0to1CUDAKernel(
    hipStream_t stream,
    u32 surfels_size,
    const CUDABuffer_<float>& surfels,
    const CUDABuffer_<u8>& active_surfels) {
  CUDA_AUTO_TUNE_1D(
      ResetSurfelAccum0to1CUDAKernel,
      512,
      surfels_size,
      0, stream,
      /* kernel parameters */
      surfels_size,
      surfels,
      active_surfels);
  CUDA_CHECK();
}


// This function only considers the depth residual. If the descriptor residual
// is also used, it should be considered jointly.
__global__ void AccumulateSurfelPositionOptimizationCoeffsFromDepthResidualCUDAKernel(
    SurfelProjectionParameters s,
    PixelCenterUnprojector depth_unprojector,
    DepthToColorPixelCorner depth_to_color,
    float color_fx,
    float color_fy,
    hipTextureObject_t color_texture,
    CUDABuffer_<u8> active_surfels) {
  const unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < s.surfels_size) {
    if (!(active_surfels(0, surfel_index) & kSurfelActiveFlag)) {
      return;
    }
    
    SurfelProjectionResult6 r;
    if (SurfelProjectsToAssociatedPixel(surfel_index, s, &r)) {
      // --- Depth residual ---
      float3 rn = s.frame_T_global.Rotate(r.surfel_normal);
      
      float depth_residual_inv_stddev =
          ComputeDepthResidualInvStddevEstimate(depth_unprojector.nx(r.px), depth_unprojector.ny(r.py), r.pixel_calibrated_depth, rn, s.depth_params.baseline_fx);
      
      const float depth_jacobian = -depth_residual_inv_stddev;
      
      float3 local_unproj;
      float raw_depth_residual;
      ComputeRawDepthResidual(
          depth_unprojector, r.px, r.py, r.pixel_calibrated_depth,
          depth_residual_inv_stddev,
          r.surfel_local_position, rn, &local_unproj, &raw_depth_residual);
      
      // Accumulate:
      // kSurfelAccum0: H
      // kSurfelAccum1: b
      const float depth_weight = ComputeDepthResidualWeight(raw_depth_residual);
      float weighted_jacobian = depth_weight * depth_jacobian;
      
      s.surfels(kSurfelAccum0, surfel_index) += weighted_jacobian * depth_jacobian;
      s.surfels(kSurfelAccum1, surfel_index) += weighted_jacobian * raw_depth_residual;
    }
  }
}

void CallAccumulateSurfelPositionOptimizationCoeffsFromDepthResidualCUDAKernel(
    hipStream_t stream,
    SurfelProjectionParameters s,
    PixelCenterUnprojector depth_unprojector,
    DepthToColorPixelCorner depth_to_color,
    float color_fx,
    float color_fy,
    hipTextureObject_t color_texture,
    CUDABuffer_<u8> active_surfels) {
  CUDA_AUTO_TUNE_1D(
      AccumulateSurfelPositionOptimizationCoeffsFromDepthResidualCUDAKernel,
      512,
      s.surfels_size,
      0, stream,
      /* kernel parameters */
      s,
      depth_unprojector,
      depth_to_color,
      color_fx,
      color_fy,
      color_texture,
      active_surfels);
  CUDA_CHECK();
}


__global__ void UpdateSurfelPositionCUDAKernel(
    u32 surfels_size,
    CUDABuffer_<float> surfels,
    CUDABuffer_<u8> active_surfels) {
  const unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfels_size) {
    if (!(active_surfels(0, surfel_index) & kSurfelActiveFlag)) {
      return;
    }
    
    float H = surfels(kSurfelAccum0, surfel_index);
    constexpr float kEpsilon = 1e-6f;
    if (H > kEpsilon) {
      float3 global_position = SurfelGetPosition(surfels, surfel_index);
      float t = -1.f * surfels(kSurfelAccum1, surfel_index) / H;
      float3 surfel_normal = SurfelGetNormal(surfels, surfel_index);
      SurfelSetPosition(&surfels, surfel_index, global_position + t * surfel_normal);
    }
  }
}

void CallUpdateSurfelPositionCUDAKernel(
    hipStream_t stream,
    u32 surfels_size,
    CUDABuffer_<float> surfels,
    CUDABuffer_<u8> active_surfels) {
  CUDA_AUTO_TUNE_1D(
      UpdateSurfelPositionCUDAKernel,
      512,
      surfels_size,
      0, stream,
      /* kernel parameters */  
      surfels_size,
      surfels,
      active_surfels);
  CUDA_CHECK();
}


__global__ void AccumulateSurfelNormalOptimizationCoeffsCUDAKernel(
    SurfelProjectionParameters s,
    CUDAMatrix3x3 global_R_frame,
    CUDABuffer_<u8> active_surfels) {
  const unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < s.surfels_size) {
    if (!(active_surfels(0, surfel_index) & kSurfelActiveFlag)) {
      return;
    }
    
    SurfelProjectionResultXY r;
    if (SurfelProjectsToAssociatedPixel(surfel_index, s, &r)) {
      // Transform the frame's normal to global space.
      float3 local_normal = U16ToImageSpaceNormal(s.normals_buffer(r.py, r.px));
      float3 global_normal = global_R_frame * local_normal;
      
      // Accumulate.
      // kSurfelAccum0: normal.x
      // kSurfelAccum1: normal.y
      // kSurfelAccum2: normal.z
      // kSurfelAccum3: count
      // NOTE: This does a simple averaging of the normals, it does not
      //       optimize according to the cost function.
      s.surfels(kSurfelAccum0, surfel_index) += global_normal.x;
      s.surfels(kSurfelAccum1, surfel_index) += global_normal.y;
      s.surfels(kSurfelAccum2, surfel_index) += global_normal.z;
      s.surfels(kSurfelAccum3, surfel_index) += 1.f;
    }
  }
}

void CallAccumulateSurfelNormalOptimizationCoeffsCUDAKernel(
    hipStream_t stream,
    SurfelProjectionParameters s,
    CUDAMatrix3x3 global_R_frame,
    CUDABuffer_<u8> active_surfels) {
  CUDA_AUTO_TUNE_1D(
      AccumulateSurfelNormalOptimizationCoeffsCUDAKernel,
      512,
      s.surfels_size,
      0, stream,
      /* kernel parameters */
      s,
      global_R_frame,
      active_surfels);
  CUDA_CHECK();
}


__global__ void UpdateSurfelNormalCUDAKernel(
    u32 surfels_size,
    CUDABuffer_<float> surfels,
    CUDABuffer_<u8> active_surfels) {
  const unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfels_size) {
    if (!(active_surfels(0, surfel_index) & kSurfelActiveFlag)) {
      return;
    }
    
    float count = surfels(kSurfelAccum3, surfel_index);
    if (count >= 1) {
      float3 normal_sum =
          make_float3(surfels(kSurfelAccum0, surfel_index),
                      surfels(kSurfelAccum1, surfel_index),
                      surfels(kSurfelAccum2, surfel_index));
      SurfelSetNormal(&surfels, surfel_index, (1.f / count) * normal_sum);
    }
  }
}

void CallUpdateSurfelNormalCUDAKernel(
    hipStream_t stream,
    u32 surfels_size,
    CUDABuffer_<float> surfels,
    CUDABuffer_<u8> active_surfels) {
  CUDA_AUTO_TUNE_1D(
      UpdateSurfelNormalCUDAKernel,
      512,
      surfels_size,
      0, stream,
      /* kernel parameters */
      surfels_size,
      surfels,
      active_surfels);
  CUDA_CHECK();
}

}
